#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <open3d/3rdparty/Eigen/Core>
#include <open3d/3rdparty/Eigen/Geometry>
#include "mcd.cuh"

__device__ void support_function_kernel(Eigen::Vector3d *vertices,
                                        int vertices_size,
                                        const Eigen::Vector3d &direction,
                                        long *shared_support_idx) {
    if (threadIdx.x == 0) {
        *shared_support_idx = 0;
        for (int i = 1; i < vertices_size; i++) {
            if (vertices[i].dot(direction) > vertices[*shared_support_idx].dot(direction)) {
                *shared_support_idx = i;
            }
        }
    }
}
//__device__ void support_function_kernel(Eigen::Vector3d *vertices,
//                                        int vertices_size,
//                                        const Eigen::Vector3d &direction,
//                                        long *shared_support_idx) {
//    long range_num = vertices_size / blockDim.x + 1;
//    long range_start = threadIdx.x * range_num;
//    if (range_start < vertices_size) {
//        long support_idx = range_start;
//        double support_value = vertices[support_idx].dot(direction);
//
//        for (long i = range_start + 1; i < range_start + range_num && i < vertices_size; i++) {
//            double support_value_new = vertices[i].dot(direction);
//            if (support_value_new > support_value) {
//                support_value = support_value_new;
//                support_idx = i;
//            }
//        }
//
//        long old_support_idx, assumed_support_idx;
//        do {
//            assumed_support_idx = *shared_support_idx;
//            if (support_value > vertices[assumed_support_idx].dot(direction)) {
//                old_support_idx = atomicCAS((unsigned long long int *) shared_support_idx,
//                                            (unsigned long long int) assumed_support_idx,
//                                            (unsigned long long int) support_idx);
//            } else {
//                break;
//            }
//        } while (old_support_idx != assumed_support_idx);
//    }
//
//    __syncthreads();
//}

__device__ void cross(const double a[3], const double b[3], double c[3]) {
    c[0] = a[1] * b[2] - a[2] * b[1];
    c[1] = a[2] * b[0] - a[0] * b[2];
    c[2] = a[0] * b[1] - a[1] * b[0];
}

__device__ void simplex_origin_lambda(Eigen::Vector3d *vertices1,
                                      Eigen::Vector3d *vertices2,
                                      const long *simplex_left,
                                      const long *simplex_right,
                                      double *simplex_lambda) {
    __shared__ double vects[5][3]; // 4 points + origin
    __shared__ double diffs[25][3];     // C(5, 2) = 20 diffs
    __shared__ double dots[25][25];
    __shared__ double crosses[25][25][3];
    __shared__ double areasqs[25][25];
    double tol = 1e-6;
    bool mainthread = threadIdx.x == 0;

    int vect2diff_map[25][2] = {
            {0, 0},
            {0, 1},
            {0, 2},
            {0, 3},
            {0, 4},
            {1, 0},
            {1, 1},
            {1, 2},
            {1, 3},
            {1, 4},
            {2, 0},
            {2, 1},
            {2, 2},
            {2, 3},
            {2, 4},
            {3, 0},
            {3, 1},
            {3, 2},
            {3, 3},
            {3, 4},
            {4, 0},
            {4, 1},
            {4, 2},
            {4, 3},
            {4, 4}
    };

    int diff2vect_map[5][5] = {
            {0,  1,  2,  3,  4},
            {5,  6,  7,  8,  9},
            {10, 11, 12, 13, 14},
            {15, 16, 17, 18, 19},
            {20, 21, 22, 23, 24}
    };

    int lambdamap[4][3] = {
            {1, 2, 3},
            {0, 2, 3},
            {0, 1, 3},
            {0, 1, 2}
    };

    // Precompute
    //// Precompute vects (basically c-space points)
    if (mainthread) {
        for (int i = 0; i < 5; i++) {
            vects[i][0] = 0;
            vects[i][1] = 0;
            vects[i][2] = 0;
        }
        for (int i = 0; i < 4; i++) {
            vects[i][0] = vertices1[simplex_left[i]][0] - vertices2[simplex_right[i]][0];
            vects[i][1] = vertices1[simplex_left[i]][1] - vertices2[simplex_right[i]][1];
            vects[i][2] = vertices1[simplex_left[i]][2] - vertices2[simplex_right[i]][2];
        }
    }

    __syncthreads();

    //// Precompute diffs (c-space edges)
    if (threadIdx.x < 25) {
        diffs[threadIdx.x][0] = vects[vect2diff_map[threadIdx.x][1]][0] - vects[vect2diff_map[threadIdx.x][0]][0];
        diffs[threadIdx.x][1] = vects[vect2diff_map[threadIdx.x][1]][1] - vects[vect2diff_map[threadIdx.x][0]][1];
        diffs[threadIdx.x][2] = vects[vect2diff_map[threadIdx.x][1]][2] - vects[vect2diff_map[threadIdx.x][0]][2];
    }

    __syncthreads();

    //// Precompute dots (c-space dot products)
    if (threadIdx.x < 25) {
        int i = threadIdx.x;
        for (int j = 0; j < 25; j++) {
            dots[i][j] = diffs[i][0] * diffs[j][0] + diffs[i][1] * diffs[j][1] + diffs[i][2] * diffs[j][2];
        }
    }

    __syncthreads();

    //// Precompute crosses & areas (c-space cross products)
    if (threadIdx.x < 25) {
        int i = threadIdx.x;
        for (int j = 0; j < 25; j++) {
            cross(diffs[i], diffs[j], crosses[i][j]);
            areasqs[i][j] = crosses[i][j][0] * crosses[i][j][0] + crosses[i][j][1] * crosses[i][j][1] +
                            crosses[i][j][2] * crosses[i][j][2];
        }
    }

    __syncthreads();

    int cardinality;

//    // detect degenerate cases
//    cardinality = simplex_lambda[0] > 0.0 + simplex_lambda[1] > 0.0 + simplex_lambda[2] > 0.0 + simplex_lambda[3] > 0.0;
//    if (cardinality == 4) {
//        if (mainthread) {
//            Eigen::Map<Eigen::Vector3d> v12x13(crosses[diff2vect_map[0][1]][diff2vect_map[0][2]]);
//            Eigen::Map<Eigen::Vector3d> v14(diffs[diff2vect_map[0][3]]);
//            bool branch = ::abs(v14.dot(v12x13)) < tol;
//            if (branch) {
//                double areas[4] = {areasqs[diff2vect_map[1][2]][diff2vect_map[1][3]],
//                                   areasqs[diff2vect_map[2][3]][diff2vect_map[2][0]],
//                                   areasqs[diff2vect_map[3][0]][diff2vect_map[3][1]],
//                                   areasqs[diff2vect_map[0][1]][diff2vect_map[0][2]]};
//                simplex_lambda[areas[0] > areas[1] ? areas[0] > areas[2] ? areas[0] > areas[3] ? 0 : 3 : 2
//                                                   : areas[1] > areas[2] ? areas[1] > areas[3] ? 1 : 3 : 2] = 0.0;
//            }
//        }
//    }
//
//    __syncthreads();
//    // calculate barycentric coordinates
//    cardinality = simplex_lambda[0] > tol + simplex_lambda[1] > tol + simplex_lambda[2] > tol +
//                                                                                          simplex_lambda[3] > tol;
//    if (cardinality == 4 && (threadIdx.x < 4)) {
//        Eigen::Map<Eigen::Vector3d> vap(diffs[diff2vect_map[0][4]]);
//        Eigen::Map<Eigen::Vector3d> vbp(diffs[diff2vect_map[1][4]]);
//
//        Eigen::Map<Eigen::Vector3d> vab(diffs[diff2vect_map[0][1]]);
//        Eigen::Map<Eigen::Vector3d> vac(diffs[diff2vect_map[0][2]]);
//        Eigen::Map<Eigen::Vector3d> vad(diffs[diff2vect_map[0][3]]);
//
//        Eigen::Map<Eigen::Vector3d> vbc(diffs[diff2vect_map[1][2]]);
//        Eigen::Map<Eigen::Vector3d> vbd(diffs[diff2vect_map[1][3]]);
//
//        Eigen::Map<Eigen::Vector3d> vbdxbc(crosses[diff2vect_map[1][3]][diff2vect_map[1][2]]);
//        Eigen::Map<Eigen::Vector3d> vacxad(crosses[diff2vect_map[0][2]][diff2vect_map[0][3]]);
//        Eigen::Map<Eigen::Vector3d> vadxab(crosses[diff2vect_map[0][3]][diff2vect_map[0][1]]);
//        Eigen::Map<Eigen::Vector3d> vabxac(crosses[diff2vect_map[0][1]][diff2vect_map[0][2]]);
//
//        Eigen::Vector3d params[4][2] = {
//                {vbp, vbdxbc},
//                {vap, vacxad},
//                {vap, vadxab},
//                {vap, vabxac}
//        };
//        double v6 = 1 / vab.dot(vabxac);
////        simplex_lambda[threadIdx.x] = v6 * (params[threadIdx.x][0].dot(params[threadIdx.x][1]));
//        if (mainthread) {
//            simplex_lambda[0] = 0.0;
//            simplex_lambda[1] = 0.0;
//            simplex_lambda[2] = 0.0;
//            simplex_lambda[3] = 0.0;
//        }
//    }
//
//    __syncthreads();

    // detect degenerate cases
    cardinality = (simplex_lambda[0] > tol) + (simplex_lambda[1] > tol) + (simplex_lambda[2] > tol) +
                  (simplex_lambda[3] > tol);
    if (cardinality == 3) {
        int ignored = simplex_lambda[0] < tol ? 0 : simplex_lambda[1] < tol ? 1 : simplex_lambda[2] < tol ? 2 : 3;
        if (mainthread) {
            bool branch =
                    areasqs[diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][1]]][diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][2]]] <
                    tol;
            if (branch) {
                double sqrtlengths[3] = {
                        dots[diff2vect_map[lambdamap[ignored][1]][lambdamap[ignored][2]]][diff2vect_map[lambdamap[ignored][1]][lambdamap[ignored][2]]],
                        dots[diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][2]]][diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][2]]],
                        dots[diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][1]]][diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][1]]]
                };
                simplex_lambda[sqrtlengths[0] > sqrtlengths[1] ? sqrtlengths[0] > sqrtlengths[2] ? lambdamap[ignored][0]
                                                                                                 : lambdamap[ignored][2]
                                                               : sqrtlengths[1] > sqrtlengths[2] ? lambdamap[ignored][1]
                                                                                                 : lambdamap[ignored][2]] = 0.0;
            }
        }
    }

    __syncthreads();

    // calculate barycentric coordinates
    cardinality = (simplex_lambda[0] > tol) + (simplex_lambda[1] > tol) + (simplex_lambda[2] > tol) +
                  (simplex_lambda[3] > tol);
    if (cardinality == 3) {
        if (mainthread) {
            int ignored =
                    simplex_lambda[0] < tol ? 0 : simplex_lambda[1] < tol ? 1 : simplex_lambda[2] < tol ? 2 : 3;
            int v0 = diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][1]];
            int v1 = diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][2]];
            int v2 = diff2vect_map[lambdamap[ignored][0]][4];

            double d00 = dots[v0][v0];
            double d01 = dots[v0][v1];
            double d11 = dots[v1][v1];
            double d20 = dots[v2][v0];
            double d21 = dots[v2][v1];
            double denom = d00 * d11 - d01 * d01;
//
//            v0 = diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][1]];
//            v1 = diff2vect_map[lambdamap[ignored][0]][lambdamap[ignored][2]];
//            v2 = diff2vect_map[lambdamap[ignored][0]][4];
//            d00 = Eigen::Map<Eigen::Vector3d>(diffs[v0]).dot(Eigen::Map<Eigen::Vector3d>(diffs[v0]));
//            d01 = Eigen::Map<Eigen::Vector3d>(diffs[v0]).dot(Eigen::Map<Eigen::Vector3d>(diffs[v1]));
//            d11 = Eigen::Map<Eigen::Vector3d>(diffs[v1]).dot(Eigen::Map<Eigen::Vector3d>(diffs[v1]));
//            d20 = Eigen::Map<Eigen::Vector3d>(diffs[v2]).dot(Eigen::Map<Eigen::Vector3d>(diffs[v0]));
//            d21 = Eigen::Map<Eigen::Vector3d>(diffs[v2]).dot(Eigen::Map<Eigen::Vector3d>(diffs[v1]));
//            denom = d00 * d11 - d01 * d01;

            double v = (d11 * d20 - d01 * d21) / denom;
            double w = (d00 * d21 - d01 * d20) / denom;
            double u = 1.0 - v - w;



//            Eigen::Vector3d a = vertices[0];
//            Eigen::Vector3d b = vertices[1];
//            Eigen::Vector3d c = vertices[2];
//            Eigen::Vector3d v0 = b - a;
//            Eigen::Vector3d v1 = c - a;
//            Eigen::Vector3d v2 = point - a;
//            double d00 = v0.dot(v0);
//            double d01 = v0.dot(v1);
//            double d11 = v1.dot(v1);
//            double d20 = v2.dot(v0);
//            double d21 = v2.dot(v1);
//            double denom = d00 * d11 - d01 * d01;
//            double v = (d11 * d20 - d01 * d21) / denom;
//            double w = (d00 * d21 - d01 * d20) / denom;
//            double u = 1.0 - v - w;

//            w = 1.0;
//            u = -2.0;
//            v = 1.0;
////            v = v < 0.0 ? 1.0 : v;
////            u = u < 0.0 ? 1.0 : u;

            simplex_lambda[lambdamap[ignored][0]] = u;
            simplex_lambda[lambdamap[ignored][1]] = v;
            simplex_lambda[lambdamap[ignored][2]] = w;
            simplex_lambda[ignored] = 0.0;
        }
    }

    __syncthreads();

    // detect degenerate cases
    cardinality = (simplex_lambda[0] > tol) + (simplex_lambda[1] > tol) + (simplex_lambda[2] > tol) +
                  (simplex_lambda[3] > tol);
    if (cardinality == 2) {
        if (mainthread) {
            int v1 = simplex_lambda[0] > 0.0 ? 0 : simplex_lambda[1] > 0.0 ? 1 : simplex_lambda[2] > 0.0 ? 2
                                                                                                         : 3;
            int v2 = simplex_lambda[3] > 0.0 ? 3 : simplex_lambda[2] > 0.0 ? 2 : simplex_lambda[1] > 0.0 ? 1
                                                                                                         : 0;
            // check two point too close below tolerance
            bool branch = dots[diff2vect_map[v1][v2]][diff2vect_map[v1][v2]] < tol;
            if (branch) simplex_lambda[v2] = 0.0;
        }
    }

    __syncthreads();

    cardinality = (simplex_lambda[0] > tol) + (simplex_lambda[1] > tol) + (simplex_lambda[2] > tol) +
                  (simplex_lambda[3] > tol);
    if (cardinality == 2) {
        if (mainthread) {
            int v1 = simplex_lambda[0] > tol ? 0 : simplex_lambda[1] > tol ? 1 : simplex_lambda[2] > tol ? 2
                                                                                                         : 3;
            int v2 = simplex_lambda[3] > tol ? 3 : simplex_lambda[2] > tol ? 2 : simplex_lambda[1] > tol ? 1 : 0;

            double u = dots[diff2vect_map[v1][4]][diff2vect_map[v1][v2]] / dots[diff2vect_map[v1][v2]][
                    diff2vect_map[v1][v2]];
            u = Eigen::Map<Eigen::Vector3d>(diffs[diff2vect_map[v1][4]]).dot(
                    Eigen::Map<Eigen::Vector3d>(diffs[diff2vect_map[v1][v2]])) /
                Eigen::Map<Eigen::Vector3d>(diffs[diff2vect_map[v1][v2]]).dot(
                        Eigen::Map<Eigen::Vector3d>(diffs[diff2vect_map[v1][v2]]));
            double v = 1.0 - u;

            simplex_lambda[0] = 0.0;
            simplex_lambda[1] = 0.0;
            simplex_lambda[2] = 0.0;
            simplex_lambda[3] = 0.0;
            simplex_lambda[v1] = v;
            simplex_lambda[v2] = u;
        }
    }

    __syncthreads();

    cardinality = (simplex_lambda[0] > tol) + (simplex_lambda[1] > tol) + (simplex_lambda[2] > tol) +
                  (simplex_lambda[3] > tol);
    if (cardinality == 1) {
        int v1 = simplex_lambda[0] > tol ? 0 : simplex_lambda[1] > tol ? 1 : simplex_lambda[2] > tol ? 2
                                                                                                     : 3;
        if (mainthread) {
            for (int i = 0; i < 4; ++i) {
                simplex_lambda[i] = i == v1 ? 1.0 : 0.0;
            }
        }
    }
    __syncthreads();
}

__global__ void mcd_kernel(Eigen::Vector3d *vertices1_gpu,
                           const int *vertices1_gpu_size,
                           Eigen::Vector3d *vertices2_gpu,
                           const int *vertices2_gpu_size,
                           Eigen::Vector3d *point1_gpu,
                           Eigen::Vector3d *point2_gpu,
                           bool *collide_gpu,
                           const double *eps) {
    __shared__ long s1;
    __shared__ long s2;
    __shared__ double d[3];
    __shared__ long simplex_left[4];
    __shared__ long simplex_right[4];
    __shared__ double simplex_lambda[4];
    __shared__ bool branch;

    bool mainthread = threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0;


    if (mainthread) {
        *collide_gpu = false;
        branch = false;
        d[0] = 0.0;
        d[1] = 0.0;
        d[2] = 1.0;
    }

    __syncthreads();

    support_function_kernel(vertices1_gpu, *vertices1_gpu_size, Eigen::Map<Eigen::Vector3d>(d), &s1);
    support_function_kernel(vertices2_gpu, *vertices2_gpu_size, -Eigen::Map<Eigen::Vector3d>(d), &s2);

    __syncthreads();

    if (mainthread) {
        simplex_left[0] = s1;
        simplex_left[1] = -1;
        simplex_left[2] = -1;
        simplex_left[3] = -1;
        simplex_right[0] = s2;
        simplex_right[1] = -1;
        simplex_right[2] = -1;
        simplex_right[3] = -1;
        simplex_lambda[0] = 1.0;
        simplex_lambda[1] = 0.0;
        simplex_lambda[2] = 0.0;
        simplex_lambda[3] = 0.0;
    }

    __syncthreads();

    Eigen::Vector3d point1_, point2_;
    double dist = (vertices1_gpu[simplex_left[0]] - vertices2_gpu[simplex_right[0]]).squaredNorm();
    bool init = true;
//    while (1) {
    for (int i = 0; i < 3; ++i) {
        simplex_origin_lambda(vertices1_gpu, vertices2_gpu, simplex_left, simplex_right, simplex_lambda);
//        if (simplex_lambda[0] > 0.0 && simplex_lambda[1] > 0.0 && simplex_lambda[2] > 0.0 && simplex_lambda[3] > 0.0) {
//            if (mainthread) {
//                *collide_gpu = true;
//            }
//            break;
//        }
        __syncthreads();

        // Get the support point
        if (mainthread) {
            point1_ =
                    simplex_lambda[0] * vertices1_gpu[simplex_left[0]] +
                    simplex_lambda[1] * vertices1_gpu[simplex_left[1]] +
                    simplex_lambda[2] * vertices1_gpu[simplex_left[2]] +
                    simplex_lambda[3] * vertices1_gpu[simplex_left[3]];
            point2_ =
                    simplex_lambda[0] * vertices2_gpu[simplex_right[0]] +
                    simplex_lambda[1] * vertices2_gpu[simplex_right[1]] +
                    simplex_lambda[2] * vertices2_gpu[simplex_right[2]] +
                    simplex_lambda[3] * vertices2_gpu[simplex_right[3]];

            d[0] = point1_[0] - point2_[0];
            d[1] = point1_[1] - point2_[1];
            d[2] = point1_[2] - point2_[2];

            double newdist = (point1_ - point2_).squaredNorm();
            if (!init && dist - newdist < *eps) {
//                branch = true;
            } else {
                dist = newdist;
            }
        }
        __syncthreads();
        if (branch) {
            break;
        }

//        if (i !=2 ){
            support_function_kernel(vertices1_gpu, *vertices1_gpu_size, -Eigen::Map<Eigen::Vector3d>(d), &s1);
            support_function_kernel(vertices2_gpu, *vertices2_gpu_size, Eigen::Map<Eigen::Vector3d>(d), &s2);

//        }

        __syncthreads();

        if (mainthread) {
//            long skip = ((simplex_lambda[0] > 0.0) +
//                         (simplex_lambda[1] > 0.0) +
//                         (simplex_lambda[2] > 0.0) +
//                         (simplex_lambda[3] > 0.0)) == 3 ? 1 : 0;
//            if (skip) {
//                for (double &k: simplex_lambda) {
//                    if (k > 0.0) {
//                        k = 0.0;
//                        break;
//                    }
//                }
//            }
//            int j = 0;
//            for (int k = 0; k < 4; ++k) {
//                if (simplex_lambda[k] > 0.0) {
//                    simplex_left[j] = simplex_left[k];
//                    simplex_right[j] = simplex_right[k];
//                    simplex_lambda[j] = simplex_lambda[k];
//                    ++j;
//                }
//            }
//            simplex_left[j] = s1;
//            simplex_right[j] = s2;
//            simplex_lambda[j] = 1.0;
//            ++j;
//            for (; j < 4; ++j) {
//                simplex_left[j] = -1;
//                simplex_right[j] = -1;
//                simplex_lambda[j] = 0.0;
//            }

            if (i < 3) {
                simplex_left[i + 1] = s1;
                simplex_right[i + 1] = s2;
                simplex_lambda[i + 1] = 1.0;
            }

            *point1_gpu = point1_;
            *point2_gpu = point2_;
        }
        init = false;
        __syncthreads();
    }
}

void mcd_cuda(std::vector<Eigen::Vector3d> &vertices1,
              std::vector<std::vector<int>> &adjacency_list1,
              std::vector<Eigen::Vector3d> &vertices2,
              std::vector<std::vector<int>> &adjacency_list2,
              Eigen::Vector3d &point1,
              Eigen::Vector3d &point2,
              bool &collide,
              double eps) {
    long vertices1_size = vertices1.size();
    long vertices2_size = vertices2.size();

    // Copy data to GPU
    Eigen::Vector3d *vertices1_gpu;
    int *vertices1_gpu_size;
    Eigen::Vector3d *vertices2_gpu;
    int *vertices2_gpu_size;
    Eigen::Vector3d *point1_gpu;
    Eigen::Vector3d *point2_gpu;
    bool *collide_gpu;
    double *eps_gpu;

    hipMalloc((void **) &vertices1_gpu, vertices1.size() * sizeof(Eigen::Vector3d));
    hipMalloc((void **) &vertices1_gpu_size, sizeof(long));
    hipMalloc((void **) &vertices2_gpu, vertices2.size() * sizeof(Eigen::Vector3d));
    hipMalloc((void **) &vertices2_gpu_size, sizeof(long));
    hipMalloc((void **) &point1_gpu, sizeof(Eigen::Vector3d));
    hipMalloc((void **) &point2_gpu, sizeof(Eigen::Vector3d));
    hipMalloc((void **) &collide_gpu, sizeof(bool));
    hipMalloc((void **) &eps_gpu, sizeof(double));

    hipMemcpy(vertices1_gpu, vertices1.data(), vertices1.size() * sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(vertices1_gpu_size, &vertices1_size, sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(vertices2_gpu, vertices2.data(), vertices2.size() * sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(vertices2_gpu_size, &vertices2_size, sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(point1_gpu, &point1, sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(point2_gpu, &point2, sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(collide_gpu, &collide, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(eps_gpu, &eps, sizeof(double), hipMemcpyHostToDevice);

    // Run kernel
    dim3 block(1024, 1, 1);
    mcd_kernel<<<1, block>>>(vertices1_gpu, vertices1_gpu_size,
                             vertices2_gpu, vertices2_gpu_size,
                             point1_gpu, point2_gpu, collide_gpu, eps_gpu);

    hipDeviceSynchronize();
    // Copy data back to CPU
    hipMemcpy(&point1, point1_gpu, sizeof(Eigen::Vector3d), hipMemcpyDeviceToHost);
    hipMemcpy(&point2, point2_gpu, sizeof(Eigen::Vector3d), hipMemcpyDeviceToHost);
    hipMemcpy(&collide, collide_gpu, sizeof(bool), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(vertices1_gpu);
    hipFree(vertices2_gpu);
    hipFree(point1_gpu);
    hipFree(point2_gpu);
    hipFree(collide_gpu);
    hipFree(eps_gpu);


}

void mcd_cpu(std::vector<Eigen::Vector3d> &vertices1,
             std::vector<std::vector<int>> &adjacency_list1,
             std::vector<Eigen::Vector3d> &vertices2,
             std::vector<std::vector<int>> &adjacency_list2,
             Eigen::Vector3d &point1,
             Eigen::Vector3d &point2,
             bool &collide,
             double eps) {
    int s1 = -1;
    int s2 = -1;
    collide = false;

    Simplex simplex;
    s1 = support_function(vertices1, adjacency_list1, Eigen::Vector3d(0, 0, 1), s1);
    s2 = support_function(vertices2, adjacency_list2, -1 * Eigen::Vector3d(0, 0, 1), s2);
    simplex.emplace_back(s1, s2, 1.0);

    double dist = (point1 - point2).squaredNorm();
    bool init = true;
    for (int i = 0; i < 3; ++i) {
        // Get the next direction
        simplex_origin_lambda(vertices1, vertices2, simplex);
        if (simplex.size() == 4 &&
            std::all_of(simplex.begin(), simplex.end(), [](auto &j) { return std::get<2>(j) > 0; })) {
            collide = true;
            break;
        }

        // Get the support point
        Eigen::Vector3d point1_ = Eigen::Vector3d::Zero();
        Eigen::Vector3d point2_ = Eigen::Vector3d::Zero();
        for (auto &j: simplex) {
            point1_ += std::get<2>(j) * vertices1[std::get<0>(j)];
            point2_ += std::get<2>(j) * vertices2[std::get<1>(j)];
        }
        auto d = point1_ - point2_;
        double newdist = d.squaredNorm();
        if (!init && dist - newdist < eps) {
            break;
        } else {
            dist = newdist;
        }
        s1 = support_function(vertices1, adjacency_list1, -d, s1);
        s2 = support_function(vertices2, adjacency_list2, d, s2);
        simplex.emplace_back(s1, s2, 1.0);

        point1 = point1_;
        point2 = point2_;
        init = false;
    }
}

int support_function(std::vector<Eigen::Vector3d> &vertices,
                     std::vector<std::vector<int>> &adjacency_list,
                     const Eigen::Vector3d &direction,
                     int start_vertex) {
    int support_index;

    if (start_vertex >= 0) {
        // Perform hill climbing
        support_index = start_vertex;
        double support_value = vertices[support_index].dot(direction);
        bool improved = true;

        while (improved) {
            improved = false;
            const std::vector<int> &adj_vertices = adjacency_list[support_index];

            int support_index_new = support_index;
            double support_value_new = support_value;

            for (int adj_vertex: adj_vertices) {
                double adj_value = vertices[adj_vertex].dot(direction);
                if (adj_value > support_value_new) {
                    support_index_new = adj_vertex;
                    support_value_new = adj_value;
                    improved = true;
                }
            }

            if (improved) {
                support_index = support_index_new;
                support_value = support_value_new;
            }
        }
    } else {
        // Perform brute force search
        support_index = std::distance(vertices.begin(),
                                      std::max_element(vertices.begin(),
                                                       vertices.end(),
                                                       [&direction](const Eigen::Vector3d &a,
                                                                    const Eigen::Vector3d &b) {
                                                           return a.dot(direction) < b.dot(direction);
                                                       }));
    }
    return support_index;
}

std::vector<double> barycentric(std::vector<Eigen::Vector3d> &vertices,
                                Eigen::Vector3d &point) {
    if (vertices.size() == 1) {
        return std::vector<double>{1};
    } else if (vertices.size() == 2) {
        Eigen::Vector3d a = vertices[0];
        Eigen::Vector3d b = vertices[1];
        Eigen::Vector3d ab = b - a;
        Eigen::Vector3d ap = point - a;
        double u = ap.dot(ab) / ab.dot(ab);
        return std::vector<double>{1 - u, u};
    } else if (vertices.size() == 3) {
        Eigen::Vector3d a = vertices[0];
        Eigen::Vector3d b = vertices[1];
        Eigen::Vector3d c = vertices[2];
        Eigen::Vector3d v0 = b - a;
        Eigen::Vector3d v1 = c - a;
        Eigen::Vector3d v2 = point - a;
        double d00 = v0.dot(v0);
        double d01 = v0.dot(v1);
        double d11 = v1.dot(v1);
        double d20 = v2.dot(v0);
        double d21 = v2.dot(v1);
        double denom = d00 * d11 - d01 * d01;
        double v = (d11 * d20 - d01 * d21) / denom;
        double w = (d00 * d21 - d01 * d20) / denom;
        double u = 1.0 - v - w;
        return std::vector<double>{u, v, w};
    } else if (vertices.size() == 4) {
        Eigen::Vector3d a = vertices[0];
        Eigen::Vector3d b = vertices[1];
        Eigen::Vector3d c = vertices[2];
        Eigen::Vector3d d = vertices[3];
        Eigen::Vector3d vap = point - a;
        Eigen::Vector3d vbp = point - b;
        Eigen::Vector3d vab = b - a;
        Eigen::Vector3d vac = c - a;
        Eigen::Vector3d vad = d - a;

        Eigen::Vector3d vbc = c - b;
        Eigen::Vector3d vbd = d - b;

        auto scalar_triple_product = [](const Eigen::Vector3d &a, const Eigen::Vector3d &b,
                                        const Eigen::Vector3d &c) {
            return a.dot(b.cross(c));
        };

        double va6 = scalar_triple_product(vbp, vbd, vbc);
        double vb6 = scalar_triple_product(vap, vac, vad);
        double vc6 = scalar_triple_product(vap, vad, vab);
        double vd6 = scalar_triple_product(vap, vab, vac);

        double v6 = 1 / scalar_triple_product(vab, vac, vad);
        return std::vector<double>{va6 * v6, vb6 * v6, vc6 * v6, vd6 * v6};
    }
}

void simplex_origin_lambda(std::vector<Eigen::Vector3d> &vertices1,
                           std::vector<Eigen::Vector3d> &vertices2,
                           Simplex &simplex) {
    double tol = 1e-6;
    if (simplex.size() == 1) {
        Eigen::Vector3d p = to_c_space(vertices1, vertices2, std::get<0>(simplex[0]), std::get<1>(simplex[0]));
        if (p.norm() < tol) {
            // collision
            simplex.clear();
            return;
        }
    } else if (simplex.size() == 2) {
        Eigen::Vector3d p1 = to_c_space(vertices1, vertices2, std::get<0>(simplex[0]), std::get<1>(simplex[0]));
        Eigen::Vector3d p2 = to_c_space(vertices1, vertices2, std::get<0>(simplex[1]), std::get<1>(simplex[1]));
        if ((p1 - p2).norm() < tol) {
            simplex.pop_back();
            simplex_origin_lambda(vertices1, vertices2, simplex);
            return;
        }

        std::vector<Eigen::Vector3d> vertices = {p1, p2};
        Eigen::Vector3d zero = Eigen::Vector3d::Zero();
        std::vector<double> lmdas = barycentric(vertices, zero);
        std::vector<bool> lmdas_code = {lmdas[0] > tol, lmdas[1] > tol};

        if (lmdas_code[0] && lmdas_code[1]) {
            std::get<2>(simplex[0]) = lmdas[0];
            std::get<2>(simplex[1]) = lmdas[1];
            return;
        } else {
            Simplex new_simplex;
            for (size_t i = 0; i < 2; ++i) {
                if (lmdas_code[i]) {
                    new_simplex.push_back(simplex[i]);
                }
            }
            simplex = new_simplex;
            simplex_origin_lambda(vertices1, vertices2, simplex);
            return;
        }
    } else if (simplex.size() == 3) {
        Eigen::Vector3d p1 = to_c_space(vertices1, vertices2, std::get<0>(simplex[0]), std::get<1>(simplex[0]));
        Eigen::Vector3d p2 = to_c_space(vertices1, vertices2, std::get<0>(simplex[1]), std::get<1>(simplex[1]));
        Eigen::Vector3d p3 = to_c_space(vertices1, vertices2, std::get<0>(simplex[2]), std::get<1>(simplex[2]));

        if ((p2 - p1).cross(p3 - p1).norm() < tol) {
            double l1 = (p1 - p2).norm();
            double l2 = (p2 - p3).norm();
            double l3 = (p3 - p1).norm();
            if (l1 >= l2 && l1 >= l3) {
                simplex.pop_back();
            } else if (l2 >= l1 && l2 >= l3) {
                simplex.erase(simplex.begin());
            } else {
                simplex.erase(simplex.begin() + 1);
            }
            simplex_origin_lambda(vertices1, vertices2, simplex);
            return;
        }

        std::vector<Eigen::Vector3d> vertices = {p1, p2, p3};
        Eigen::Vector3d zero = Eigen::Vector3d::Zero();
        std::vector<double> lmdas = barycentric(vertices, zero);
        std::vector<bool> lmdas_code = {lmdas[0] > tol, lmdas[1] > tol, lmdas[2] > tol};

        if (lmdas_code[0] && lmdas_code[1] && lmdas_code[2]) {
            for (size_t i = 0; i < 3; ++i) {
                std::get<2>(simplex[i]) = lmdas[i];
            }
            return;
        } else {
            Simplex new_simplex;
            for (size_t i = 0; i < 3; ++i) {
                if (lmdas_code[i]) {
                    new_simplex.push_back(simplex[i]);
                }
            }
            simplex = new_simplex;
            simplex_origin_lambda(vertices1, vertices2, simplex);
            return;
        }
    } else if (simplex.size() == 4) {
        Eigen::Vector3d p1 = to_c_space(vertices1, vertices2, std::get<0>(simplex[0]), std::get<1>(simplex[0]));
        Eigen::Vector3d p2 = to_c_space(vertices1, vertices2, std::get<0>(simplex[1]), std::get<1>(simplex[1]));
        Eigen::Vector3d p3 = to_c_space(vertices1, vertices2, std::get<0>(simplex[2]), std::get<1>(simplex[2]));
        Eigen::Vector3d p4 = to_c_space(vertices1, vertices2, std::get<0>(simplex[3]), std::get<1>(simplex[3]));

        double volume = std::abs((p4 - p1).dot((p2 - p1).cross(p3 - p1))) / 6;

        if (volume < tol) {
            double l1 = (p2 - p1).cross(p3 - p1).norm();
            double l2 = (p3 - p2).cross(p4 - p2).norm();
            double l3 = (p4 - p3).cross(p1 - p3).norm();
            double l4 = (p1 - p4).cross(p2 - p4).norm();

            if (l1 >= l2 && l1 >= l3 && l1 >= l4) {
                simplex.pop_back();
            } else if (l2 >= l1 && l2 >= l3 && l2 >= l4) {
                simplex.erase(simplex.begin());
            } else if (l3 >= l1 && l3 >= l2 && l3 >= l4) {
                simplex.erase(simplex.begin() + 1);
            } else {
                simplex.erase(simplex.begin() + 2);
            }
            simplex_origin_lambda(vertices1, vertices2, simplex);
            return;
        }

        std::vector<Eigen::Vector3d> vertices = {p1, p2, p3, p4};
        Eigen::Vector3d zero = Eigen::Vector3d::Zero();
        std::vector<double> lmdas = barycentric(vertices, zero);
        std::vector<bool> lmdas_code = {lmdas[0] > tol, lmdas[1] > tol, lmdas[2] > tol, lmdas[3] > tol};

        if (std::all_of(lmdas_code.begin(), lmdas_code.end(), [](bool b) { return b; })) {
            return;
        }
        Simplex new_simplex;
        for (size_t i = 0; i < 4; ++i) {
            if (lmdas_code[i]) {
                new_simplex.push_back(simplex[i]);
            }
        }
        simplex = new_simplex;
        simplex_origin_lambda(vertices1, vertices2, simplex);
        return;
    } else {
        throw std::runtime_error("Simplex has more than 4 points");
    }

}